#include "hip/hip_runtime.h"
#include "../common/book.h"

#define SIZE    (100*1024*1024)
#define ELEMENTS    (SIZE / sizeof(unsigned int))
#define HASH_ENTRIES     1024

//Structure contains a key value pair as well as pointer to next entry.
//In this way it is very similar to a linked list.
struct Entry {
    unsigned int    key;
    void            *value;
    Entry           *next;
};

//Table Structure
struct Table {
    size_t  count;
    Entry   **entries;
    Entry   *pool;
    Entry   *firstFree;
};

//Here, the hash is more complex.
size_t hash( unsigned int key, size_t count ) {
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//Table initilization using CUDA memset and malloc.
void initialize_table( Table &table, int entries, int elements ) {
    table.count = entries;
    table.entries = (Entry**)calloc( entries, sizeof(Entry*) );
    table.pool = (Entry*)malloc( elements * sizeof( Entry ) );
    table.firstFree = table.pool;
}

//Free table once done timing
void free_table( Table &table ) {
    free( table.entries );
    free( table.pool );
}

//Implementation of cuckoo hash insertion
void add_to_table( Table &table, unsigned int key, void *value ) {
    size_t hashValue = hash( key, table.count );
    bool hashEmpty = false;
    while hashEmpty == false{
        Entry *location = table.firstFree++;
        location->key = key;
        location->value = value;
        location->next = table.entries[hashValue];
    }

    table.entries[hashValue] = location;
}

//Here, the table is verified to make sure all insertions were successful
void verify_table( const Table &table ) {
    int count = 0;
    for (size_t i=0; i<table.count; i++) {
        Entry   *current = table.entries[i];
        while (current != NULL) {
            ++count;
            if (hash( current->key, table.count ) != i)
                printf( "%d hashed to %ld, but was located at %ld\n", current->key, hash( current->key, table.count ), i );
            current = current->next;
        }
    }
    if (count != ELEMENTS)
        printf( "%d elements found in hash table.  Should be %ld\n", count, ELEMENTS );
    else
        printf( "All %d elements found in hash table.\n", count);
}


int main( void ) {
    unsigned int *buffer = (unsigned int*)big_random_block( SIZE );

    Table table;
    initialize_table( table, HASH_ENTRIES, ELEMENTS );
    
    //clock is used for timing.
    clock_t start, stop;
    start = clock();

    for (int i=0; i<ELEMENTS; i++) {
        add_to_table( table, buffer[i], (void*)NULL );
    }

    stop = clock();
    float   elapsedTime = (float)(stop - start) / (float)CLOCKS_PER_SEC * 1000.0f;
    printf( "Time to hash:  %3.1f ms\n", elapsedTime );


    verify_table( table );

    free_table( table );
    free( buffer );
    return 0;
}

